#include <fstream>
#include <vector>
#include <string>
#include <sstream>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;


__global__ void Dim2_Calculation(float * __restrict__ d_tem_res, float * __restrict__ d_tem_meo, 
    const float * __restrict__ d_tem_fix, const int width, const int height, const float k)
{
    const int curId = blockIdx.x * blockDim.x + threadIdx.x;
    const int w = curId/height;
    const int h = curId - w*height;
    int top, down, left, right;
    if( w<width && h<height){
        //float tmp = 0;
        if (w>0){
            //printf("(w-1)*height+h = %d\n", (w-1)*height+h);
            //tmp += k * ( - d_tem_meo[curId] + d_tem_meo[(w-1)*height+h]);
            left = w-1;
        }else{
            left=w;
        }
        if (w<width-1){
            //printf("(w+1)*height+h = %d\n", (w+1)*height+h);
            //tmp += k * ( - d_tem_meo[curId] + d_tem_meo[(w+1)*height+h]);
            right = w+1;
        }else{
            right = w;
        }
        if (h>0){
            //printf("(w)*height+h-1 = %d\n", (w)*height+h-1);
            //tmp += k * ( - d_tem_meo[curId] + d_tem_meo[w*height+(h-1)]);
            top = h-1;
        }else{
            top = h;
        }
        if (h<height-1){
            //printf("(w)*height+h+1 = %d\n", (w)*height+h+1);
            //tmp += k * ( - d_tem_meo[curId] + d_tem_meo[w*height+(h+1)]);
            down = h+1;
        }else{
            down = h;
        }
        d_tem_res[curId] = d_tem_meo[curId]+k*(d_tem_meo[left*height+h]+d_tem_meo[right*height+h]
            + d_tem_meo[w*height+top] + d_tem_meo[w*height+down] - 4*d_tem_meo[curId]);
    }
    if ( d_tem_fix[curId] != -1){
        d_tem_res[curId] = d_tem_fix[curId];
    }
    // wait until other thread if finished.
    __syncthreads();
    d_tem_meo[curId] = d_tem_res[curId];

}


__global__ void Dim3_Calculation(float * __restrict__ d_tem_res, float * __restrict__ d_tem_meo, 
    const float * __restrict__ d_tem_fix, const int width, const int height, const int depth, const float k)
{
    const int curId = blockIdx.x * blockDim.x + threadIdx.x;
    const int d = curId/(height*width);
    const int w = (curId-d*(height*width))/height;
    const int h = curId-d*(height*width) - w*height;
    int top, down, left, right, front, back;
    if( w<width && h<height && d<depth){
        if (w>0){
            left = w-1;
        }else{
            left=w;
        }
        if (w<width-1){
            right = w+1;
        }else{
            right = w;
        }
        if (h>0){
            top = h-1;
        }else{
            top = h;
        }
        if (h<height-1){
            down = h+1;
        }else{
            down = h;
        }
        if (d>0){
            front = d-1;
        }else{
            front = d;
        }
        if (d<depth-1){
            back = d+1;
        }else{
            back = d;
        }
        d_tem_res[curId] = d_tem_meo[curId]+k*(d_tem_meo[d*(height*width)+left*height+h]+d_tem_meo[d*(height*width)+right*height+h]
            + d_tem_meo[d*(height*width)+w*height+top] + d_tem_meo[d*(height*width)+w*height+down] 
            + d_tem_meo[front*(height*width)+w*height+h] + d_tem_meo[back*(height*width)+w*height+h] - 6*d_tem_meo[curId]);
        
    }
    if ( d_tem_fix[curId] != -1){
        d_tem_res[curId] = d_tem_fix[curId];
    }
    // wait until other thread if finished.
    __syncthreads();
    d_tem_meo[curId] = d_tem_res[curId];

}





int main(int argc,char**argv)
{
// ------------------------------------ initial parameter --------------------------------------
    string Dimension;
    string path = argv[1];
    ifstream cfile(path.c_str());
    int timestep, width, height, depth, totalLength;
    int location_x, location_y, location_z, fix_width, fix_height, fix_depth;
    float init_temp, ftemp, k;
    float *tem_res, *tem_meo, *tem_fix;
    float *d_tem_res, *d_tem_meo, *d_tem_fix; 

// ------------------------------------Reading the config file ---------------------------------------

    string l;
    vector<string> fileContent;
    while(getline(cfile, l))
    {
    	if(l[0] != '#' && !l.empty())
		{fileContent.push_back(l);}
	}
        
    // -------------------------Dimension, k, timestep, init_temp---------------------------
    Dimension = fileContent[0];
    if(fileContent[1][0]=='.')
    {fileContent[1].insert(fileContent[1].begin(), '0');}
    k = (float)atof(fileContent[1].c_str());
    timestep = atoi(fileContent[2].c_str());
    init_temp = (float)atof(fileContent[4].c_str());
    cout << "k=" << k << " timestep=" << timestep << " BeginTemperture=" << init_temp << endl;

// --------------------begin calculation based on Dimension----------------------
	
	// read width and height, then build the matrix.
    if(Dimension=="2D")
    {
        string::size_type pos = fileContent[3].find(",");
        width = atoi(fileContent[3].substr(0, pos).c_str());
        height = atoi(fileContent[3].substr(pos+1).c_str());
        cout << "Width=" << width << " Height=" << height << endl;
        totalLength = width*height;
    }
    else{
    	string::size_type pos = fileContent[3].find(",");
    	string::size_type pos2 = fileContent[3].find_last_of(",");
        width = atoi(fileContent[3].substr(0, pos).c_str());
        height = atoi(fileContent[3].substr(pos+1,pos2-pos-1).c_str());
        depth = atoi(fileContent[3].substr(pos2+1).c_str());
        cout << "Width=" << width << " Height=" << height << " Depth="<< depth << endl;
        totalLength = width*height*depth;
    }
    tem_res = (float *)malloc(totalLength * sizeof(float));
    tem_meo = (float *)malloc(totalLength * sizeof(float));
    tem_fix = (float *)malloc(totalLength * sizeof(float));
    for (int i = 0; i < totalLength; ++i)
    {
        tem_res[i] = init_temp;
        tem_meo[i] = init_temp;
        tem_fix[i] = -1;
    }

// ------------------------ initialize matrix ----------------------------------
    if(Dimension=="2D")
    {
        for(int i=5; i<fileContent.size(); i++)
        {
            string s = fileContent[i];
            location_x = atoi(s.substr(0,s.find(",")).c_str());
            s.erase(0,s.find(",")+1);
            location_y = atoi(s.substr(0,s.find(",")).c_str());
            s.erase(0,s.find(",")+1);
            fix_width = atoi(s.substr(0,s.find(",")).c_str());
            s.erase(0,s.find(",")+1);
            fix_height = atoi(s.substr(0,s.find(",")).c_str());
            s.erase(0,s.find(",")+1);
            ftemp = (float)atof(s.c_str());

            for(int w=location_x; w<location_x+fix_width; w++)
            {
                for(int h=location_y; h<location_y+fix_height; h++)
                {
                    tem_res[w*height+h] = ftemp;
                    tem_meo[w*height+h] = ftemp;
                    tem_fix[w*height+h] = ftemp;
                }
            }
        }
        //for (int i=0; i<totalLength; i++){cout << tem_fix[i] << endl;}
    }
    else{
    	for(int i=5; i<fileContent.size(); i++)
        {
            string s = fileContent[i];
            location_x = atoi(s.substr(0,s.find(",")).c_str());
            s.erase(0,s.find(",")+1);
            location_y = atoi(s.substr(0,s.find(",")).c_str());
            s.erase(0,s.find(",")+1);
            location_z = atoi(s.substr(0,s.find(",")).c_str());
            s.erase(0,s.find(",")+1);
            fix_width = atoi(s.substr(0,s.find(",")).c_str());
            s.erase(0,s.find(",")+1);
            fix_height = atoi(s.substr(0,s.find(",")).c_str());
            s.erase(0,s.find(",")+1);
            fix_depth = atoi(s.substr(0,s.find(",")).c_str());
            s.erase(0,s.find(",")+1);
            ftemp = (float)atof(s.c_str());

            for(int w=location_x; w<location_x+fix_width; w++)
            {
                for(int h=location_y; h<location_y+fix_height; h++)
                {
                    for(int d=location_z; d<location_z+fix_depth; d++)
                    {
                        tem_res[d*width*height + w*height + h] = ftemp;
                    	tem_meo[d*width*height + w*height + h] = ftemp;
                    	tem_fix[d*width*height + w*height + h] = ftemp;
                    }
                }
            }
        }
        //for (int i=0; i<totalLength; i++){cout << tem_fix[i] << endl;}
    }

	hipMalloc((void **)&d_tem_res, totalLength * sizeof(float));
	hipMalloc((void **)&d_tem_meo, totalLength * sizeof(float));
	hipMalloc((void **)&d_tem_fix, totalLength * sizeof(float));
    hipMemcpy(d_tem_res, tem_res, totalLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tem_meo, tem_meo, totalLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tem_fix, tem_fix, totalLength * sizeof(float), hipMemcpyHostToDevice);

// ----------------------- call Cuda Calculation -----------------------------------
    cout << "You begin Cuda Calculation" << endl;
    int bknum = 128;
    if(Dimension=="2D")
    {   
        for (int i = 0; i < timestep; i++) 
        {   
            //cout << "Cuda Calculation " << i <<endl;
        	Dim2_Calculation <<<(totalLength+bknum-1)/bknum, bknum>>>(d_tem_res, d_tem_meo, d_tem_fix, width, height, k);
        }
        hipMemcpy(tem_res, d_tem_res, totalLength*sizeof(float), hipMemcpyDeviceToHost);
    }
    else
    {
        for (int i = 0; i < timestep; i++) 
        {
        	Dim3_Calculation <<<(totalLength+bknum-1)/bknum, bknum>>>(d_tem_res, d_tem_meo, d_tem_fix, width, height, depth, k);
        }
        hipMemcpy(tem_res, d_tem_res, totalLength*sizeof(float), hipMemcpyDeviceToHost);
    }


// ------------------------- Writing into output.csv -------------------------------
    ofstream result;
    result.open("heatOutput.csv");
    if(Dimension=="2D")
    {
        for (int w = 0; w < width; w++) 
        {
            for (int h = 0; h < height; h++) 
            {	
            	result << tem_res[w*height+h];
                if(h<height-1)
                {result << ",";}
            }
            result << '\n';
        }
    }
    else
    {
        for(int d=0; d<depth; d++)
        {
            for (int w = 0; w < width; w++) 
            {
                for (int h = 0; h < height; h++) 
                {	
                	result << tem_res[d*width*height+w*height+h];
                	if(h<height-1)
                	{result << ",";}
                }
                result << '\n';
            }
            result << '\n';
        }        
    }
    result.close();

// -------------------- free meomery -------------------
    hipFree(d_tem_res); 
    hipFree(d_tem_meo); 
    hipFree(d_tem_fix); 
    free(tem_res); 
    free(tem_meo); 
    free(tem_fix); 

	return 0;
}